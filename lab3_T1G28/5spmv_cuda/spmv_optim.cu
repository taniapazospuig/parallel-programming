#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define N 1024
#define ROWSIZE 9
#define THREADS_PER_BLOCK 128


__global__ void cuspmv(int m, double* dvals, int *dcols, double* dx, double *dy)
{
    // to simplify this function uses the macro ROWSIZE
    // instead of the input argument r
    __shared__ double sdx[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int row = tid / N;
    int col = tid % N;

    double sum = 0.0;

    for (int j = 0; j < ROWSIZE; ++j) {
        int index = row * ROWSIZE + j;
        if (dcols[index] == col) {
            sdx[threadIdx.x] = dx[dcols[index]];
            __syncthreads();
            for (int k = 0; k < ROWSIZE; ++k) {
                sum += dvals[index] * sdx[k];
            }
            break;
        }
    }

    if (tid < m) {
        dy[tid] = sum;
    }
}


void spmv_cpu(int m, int r, double* vals, int* cols, double* x, double* y)
{
    for (int i=0; i<m; i++) {
        double sum = 0.0;
        for (int j=0; j<r; j++) {
            int col = cols[i * r + j];
            double val = vals[i * r + j];
            sum += val * x[col];
        }
        y[i] = sum;
    }
}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count = 0;
    for(int j = 0; j < N ; j++){
        for(int i = 0; i < N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +  j     *N;
            indx[3] = i - 1 +  j     *N;
            indx[4] = i     +  j     *N;
            indx[5] = i + 1 +  j     *N;
            indx[6] = i + 2 +  j     *N;
            indx[7] = i     + (j + 1)*N;
            indx[8] = i     + (j + 2)*N;

            for(int row = 0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    cols[row + row_count*ROWSIZE] = indx[row];
                    if(row == 4)
                    {
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{
    int vec_size = N * N;

    float time_cpu, time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* x = (double*)malloc(vec_size * sizeof(double));
    double* y_cpu = (double*)malloc(vec_size * sizeof(double));
    double* y_gpu = (double*)malloc(vec_size * sizeof(double));

    double* Avals = (double*)malloc(ROWSIZE * vec_size * sizeof(double));
    int* Acols = (int*)malloc(ROWSIZE * vec_size * sizeof(int));

    // fill vector with sinusoidal for testing the code
    for (int i = 0; i < vec_size; i++) {
        x[i] = sin(i * 0.01);
        y_cpu[i] = 0.0;
    }

    fill_matrix(Avals, Acols);

    // measure time of CPU implementation
    hipEventRecord(start);

    for (int i = 0; i < 100; ++i)
        spmv_cpu(vec_size, ROWSIZE, Avals, Acols, x, y_cpu);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cpu, start, stop);

    // complete here your cuda code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int* dAcols;

    // allocate arrays in GPU
    hipMalloc((void**)&dx, vec_size * sizeof(double));
    hipMalloc((void**)&dy_gpu, vec_size * sizeof(double));
    hipMalloc((void**)&dAvals, ROWSIZE * vec_size * sizeof(double));
    hipMalloc((void**)&dAcols, ROWSIZE * vec_size * sizeof(int));

    // transfer data to GPU
    hipMemcpy(dx, x, vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAvals, Avals, ROWSIZE * vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAcols, Acols, ROWSIZE * vec_size * sizeof(int), hipMemcpyHostToDevice);

    // calculate threads and blocks
    int numBlocks = (vec_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // create the gridBlock
    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        cuspmv<<<numBlocks, THREADS_PER_BLOCK>>>(vec_size, dAvals, dAcols, dx, dy_gpu);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);

    // transfer result to CPU RAM
    hipMemcpy(y_gpu, dy_gpu, vec_size * sizeof(double), hipMemcpyDeviceToHost);

    // free arrays in GPU
    hipFree(dx);
    hipFree(dy_gpu);
    hipFree(dAvals);
    hipFree(dAcols);

    // comparison between gpu and cpu results
    double norm2 = 0.0;
    for (int i = 0; i < vec_size; i++)
        norm2 += (y_cpu[i] - y_gpu[i]) * (y_cpu[i] - y_gpu[i]);

    norm2 = sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, size %d\n", norm2, vec_size);

    printf("CPU Time: %lf\n", time_cpu / 1000);
    printf("GPU Time: %lf\n", time_gpu / 1000);

    // free CPU arrays
    free(x);
    free(y_cpu);
    free(y_gpu);
}
