#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define N 1024
#define ROWSIZE 9
#define THREADS_PER_BLOCK 128


__global__ void cuspmv(int m, int r, double* dvals, int *dcols, double* dx, double *dy)
{


}


void spmv_cpu(int m, int r, double* vals, int* cols, double* x, double* y)
{

}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count = 0;
    for(int j = 0; j < N ; j++){
        for(int i = 0; i < N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +  j     *N;
            indx[3] = i - 1 +  j     *N;
            indx[4] = i     +  j     *N;
            indx[5] = i + 1 +  j     *N;
            indx[6] = i + 2 +  j     *N;
            indx[7] = i     + (j + 1)*N;
            indx[8] = i     + (j + 2)*N;

            for(int row = 0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    cols[row + row_count*ROWSIZE] = indx[row];
                    if(row == 4)
                    {
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{
    int vec_size = N*N;

    float time_cpu, time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* x     = (double*) malloc (vec_size*sizeof(double));
    double* y_cpu = (double*) malloc (vec_size*sizeof(double));
    double* y_gpu = (double*) malloc (vec_size*sizeof(double));

    double* Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    int*    Acols = (int   *) malloc (ROWSIZE*vec_size*sizeof(int));


    // fill vector with sinusoidal for testing the code
    for(int i = 0; i < vec_size; i++)
    {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
    }

    fill_matrix(Avals, Acols);

    // measure time of CPU implementation
    hipEventRecord(start);

    for (int i = 0; i < 100; ++i)
        spmv_cpu(vec_size, ROWSIZE, Avals, Acols, x, y_cpu);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cpu, start, stop);


    // complete here your cuda code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int*    dAcols;


    // allocate arrays in GPU

    // transfer data to GPU

    // calculate threads and blocks

    // create the gridBlock

    for( int i=0; i<100; i++){
        // call your GPU kernel here
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);

    // transfer result to CPU RAM

    // free arrays in GPU


    // comparison between gpu and cpu results
    double norm2 = 0.0;
    for(int i = 0; i < vec_size; i++)
        norm2 += (y_cpu[i] - y_gpu[i])*(y_cpu[i] - y_gpu[i]);

    norm2 = sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, size %d\n", norm2, vec_size);

    printf("CPU Time: %lf\n", time_cpu/1000);
    printf("GPU Time: %lf\n", time_gpu/1000);

    // free CPU arrays
    free(x);
    free(y_cpu);
    free(y_gpu);
    free(Acols);
    free(Avals);
}
